#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "KernelGPU.cuh"


__global__ void gpuNormAndShift(float* gpuP, int size, float mean, float variance, float gamma, float beta, float epsilon)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		gpuP[id] = gamma * ((gpuP[id] - mean) / sqrt(variance + epsilon)) + beta;
	}
}

void gpuNormalizeAndShift(struct CpuGpuMat* cgMat, float mean, float variance, float gamma, float beta, float epsilon)
{
	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(cgMat->Size) / double(threadsPerBlock));

	gpuNormAndShift << < blocksPerGrid, threadsPerBlock >> > ((float*)cgMat->gpuP, cgMat->Size, mean, variance, gamma, beta, epsilon);
}